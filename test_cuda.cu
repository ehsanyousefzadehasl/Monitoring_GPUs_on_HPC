#include<iostream>
#include<hip/hip_runtime.h>

// kernel (function) to be executed on the GPU
__global__ void helloWorld() {
        printf("Hello from thread %d from block %d\n", threadIdx.x, blockIdx.x);
}

int main() {
        // launch the GPU kernel

	helloWorld<<<1024, 1024>>>();
        
        hipDeviceSynchronize();

        return 0;
}