#include<iostream>
#include<hip/hip_runtime.h>

// kernel (function) to be executed on the GPU
__global__ void helloWorld() {
        printf("Hello from thread %d from block %d\n", threadIdx.x, blockIdx.x);
}

int main() {
        // launch the GPU kernel

        for(int i = 0; i < 100000; ++i){
            helloWorld<<<2048, 1024>>>();
        }
        
        hipDeviceSynchronize();

        return 0;
}